/*
* This file is part of the Lattice Boltzmann multiple GPU distribution. 
(https://github.com/evstigneevnm/LBM_D3Q19_mGPU).
* Copyright (c) 2017-2018 Evstigneev Nikolay Mikhaylovitch and Ryabkov Oleg Igorevich.
*
* This program is free software: you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, version 2 only.
*
* This program is distributed in the hope that it will be useful, but
* WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
* General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with this program. If not, see <http://www.gnu.org/licenses/>.
*/


#include "single_step.h"


void run_single_step(dim3 dimGrid, dim3 dimBlock, int Nx, int Ny, int Nz, communication_variables *COM, microscopic_variables MV_d1, microscopic_variables MV_d2, macroscopic_variables NV_d, control_variables CV_d, real omega, real delta)
{

       
        kernel_collide_0_18<<< dimGrid, dimBlock>>>( delta, NV_d.H, 0.0, 0.0, 0.0, Nx, Ny, Nz, omega, NV_d.ux, NV_d.uy, NV_d.uz, NV_d.rho, CV_d.bc, MV_d1, MV_d2);

        exchange_boundaries_MPI(dimGrid, dimBlock, Nx, Ny, Nz, COM, MV_d2, MV_d1);
        kernel_stream3D_0_18_forward<<< dimGrid, dimBlock>>>(Nx, Ny, Nz, CV_d.bc, MV_d2, MV_d1);

}

void run_single_step_streams(dim3 dimGrid, dim3 dimBlock, int Nx, int Ny, int Nz, communication_variables *COM, microscopic_variables MV_d1, microscopic_variables MV_d2, macroscopic_variables NV_d, control_variables CV_d, real omega, real delta)
{

    MPI_Request request_send1, request_send2, request_recv1, request_recv2;
    int tag=0;
    

    kernel_collide_0_18<<< dimGrid, dimBlock>>>( delta, NV_d.H, 0.0, 0.0, 0.0, Nx, Ny, Nz, omega, NV_d.ux, NV_d.uy, NV_d.uz, NV_d.rho, CV_d.bc, MV_d1, MV_d2);


    copy_send_buffers_streams(dimGrid, dimBlock, Nx, Ny, Nz, COM, MV_d2);

    if(COM->Face1Bufer_size>0)
    {
        MPI_Isend(COM->Face1BuferSend_device, COM->Face1Bufer_size, MPI_real, COM->Face1proc, tag, MPI_COMM_WORLD, &request_send1);        
    }   
    if(COM->Face2Bufer_size>0)
    {
        MPI_Isend(COM->Face2BuferSend_device, COM->Face2Bufer_size, MPI_real, COM->Face2proc, tag, MPI_COMM_WORLD, &request_send2);        
    }

    kernel_stream3D_0_18_forward<<< dimGrid, dimBlock, 0, COM->streams[0]>>>(Nx, Ny, Nz, CV_d.bc, MV_d2, MV_d1);
    

    if(COM->Face1Bufer_size>0)
    {
        MPI_Irecv(COM->Face1BuferRecv_device, COM->Face1Bufer_size, MPI_real, COM->Face1proc, tag, MPI_COMM_WORLD, &request_recv1);
        MPI_Wait(&request_recv1, MPI_STATUS_IGNORE);
    }   
    if(COM->Face2Bufer_size>0)
    {
        MPI_Irecv(COM->Face2BuferRecv_device , COM->Face2Bufer_size, MPI_real, COM->Face2proc, tag, MPI_COMM_WORLD, &request_recv2);
        MPI_Wait(&request_recv2, MPI_STATUS_IGNORE);
    }
    copy_recv_buffers_streams(dimGrid, dimBlock, Nx, Ny, Nz, COM, MV_d1);
    if(COM->Face1Bufer_size>0)
    {
        MPI_Wait(&request_send1, MPI_STATUS_IGNORE);
    }
    if(COM->Face2Bufer_size>0)
    {
        MPI_Wait(&request_send2, MPI_STATUS_IGNORE);
    }    
    CUDA_SAFE_CALL(hipStreamSynchronize( COM->streams[0]));


}