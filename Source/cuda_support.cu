#include "cuda_support.h"


#if __DEVICE_EMULATION__

int InitCUDA(int PCI_ID){return true;}

#else

int InitCUDA(int PCI_ID)
{

    
    int count = 0;
    int i = 0;

    hipGetDeviceCount(&count);
    if(count == 0)
    {
        fprintf(stderr, "There is no compartable device found.\n");
        return -1;
    }
    
    int deviceNumber=0;
    int deviceNumberTemp=0;
    
    if(count>1)
    {

        if(PCI_ID==-1)
        {
            for(i = 0; i < count; i++) 
            {
                hipDeviceProp_t deviceProp;
                hipGetDeviceProperties(&deviceProp, i);
                printf( "#%i:   %s, pci-bus id:%i %i %i \n", i, &deviceProp,deviceProp.pciBusID,deviceProp.pciDeviceID,deviceProp.pciDomainID);
            }            
            printf("Device number for it to use>>>\n",i);
            scanf("%i", &deviceNumberTemp);
        }
        else
        {
            hipDeviceProp_t deviceProp;
            for(int j=0;j<count;j++)
            {
                hipGetDeviceProperties(&deviceProp, j);
                if(deviceProp.pciBusID==PCI_ID)
                    deviceNumberTemp = j;
            }

            //printf("Using device number %i\n",deviceNumberTemp);
        }
        deviceNumber=deviceNumberTemp;
    
    }
    else
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, deviceNumber);
        printf( "#%i:   %s, pci-bus id:%i %i %i \n", deviceNumber, &deviceProp,deviceProp.pciBusID,deviceProp.pciDeviceID,deviceProp.pciDomainID);
        printf( "       using it...\n");    
    }

    hipSetDevice(deviceNumber);
    
    return deviceNumber;
}
#endif





// void probe_boundary()
// {




    
// }